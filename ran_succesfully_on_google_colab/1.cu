#include "hip/hip_runtime.h"
%%cu
#include <cstdio>
#include <iostream>

using namespace std;

__global__ 
void maxi(int* a, int* b, int n)
{
    int block = 256 * blockIdx.x;
    int max = 0;
 
    for (int i = block; i < min(256 + block, n); i++) {
 
        if (max < a[i]) {
            max = a[i];
        }
    }
    b[blockIdx.x] = max;
}
 

int main()
{

	int n;
	n = 3 << 2;
 cout<<n<<endl;
	int a[n];

	for (int i = 0; i < n; i++) {
		a[i] = rand() % n;
		cout << a[i] << "\t";
	}
 
 hipEvent_t start, end;
    int *ad, *bd;
    int size = n * sizeof(int);
    hipMalloc(&ad, size);
    hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
    int grids = ceil(n * 1.0f / 256.0f);
    hipMalloc(&bd, grids * sizeof(int));
 

    dim3 grid(grids, 1);
    dim3 block(1, 1);
 
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
 int c=9;
 while(c){cout<<c--;}

 while (n > 1) {
        maxi <<< grids, block >>> (ad, bd, n);
        n = ceil(n * 1.0f / 256.0f);
        hipMemcpy(ad, bd, n * sizeof(int), hipMemcpyDeviceToDevice);
    }
 
    hipEventRecord(end);
    hipEventSynchronize(end);
 

	cout << "The time3 required : ";
	cout << time << " "<<n<<endl;
}

